
#include <hip/hip_runtime.h>
/*
//#include "cuda_runtime.h"
//#include "device_launch_parameters.h"
//
//#include <stdio.h>
*/


//-------------------------------------219 Understadn the device better---------------------------------------

/* computer architectures classification
		-SISD - Single instruction single data
		-SIMD - single instruction multiple data
		-MISD - multiple instruction single data
		-MIMD - multiple instruction multiple data 
	CUDA se basa en SIMD
		-SIMT - single instruction multiple threads

*/


//------------------------------------- 220 Warps ---------------------------------------

/*
* -thread blocks are divide in to smaller units called warps each having 32 consecutive threads
* -warps can be defined asthe basic unit of execution in a SM
* -all threads in a warp are executed in single instrucction multiple thread (SIMT) fashion
* 
* -Los bloques se dividen en subbloques de 32
*	-ej.1. Un bloque de 128 se dividira en 4 bloques de 32 (0-31)(32-63)(64-95)(96-127)
*	-ej.2. Un bloque de 80 se dividira en 4 bloques de los cuales se dividiran (0-31)(32-39)(40-71)(72-79)
*	-ej.3. Un bloque de 1 thread activara un bloque de 32 threads de los cuales solo utilizara 1.
*/

// EJEMPLO DE USO DE BLOQUES CUANDO SE UTILIZAN 40 HILOS
//#include "cuda_runtime.h"
//#include "device_launch_parameters.h"
//
//#include <stdio.h>
//#include <stdlib.h>
//
//__global__ void print_details_of_warps()
//{
//	int gid = blockIdx.y * gridDim.x * blockDim.x + blockIdx.x * blockDim.x + threadIdx.x;
//
//	int warp_id = threadIdx.x / 32;
//
//	int gbid = blockIdx.y * gridDim.x + blockIdx.x;
//
//	printf( "tid: %d, bid.x: %d, bid.y: %d, gid: %d, warp_id: %d, gbid: %d \n",
//		threadIdx.x, blockIdx.x, blockIdx.y, gid, warp_id, gbid);
//}
//
//
//int main(int argc, char** argv) {
//
//	dim3 block_size(42);
//	dim3 grid_size(2, 2);
//
//	print_details_of_warps << <grid_size, block_size >> > ();
//	cudaDeviceSynchronize();
//
//	cudaDeviceReset();
//	return EXIT_SUCCESS;
//
//}

//------------------------------------- 221 Warp divergence ---------------------------------------
/*
	- forzar a algunos hilos en el warp para ejecutar diferentes instrucciones
	- la eficiencia de las ramas se puede medir para un kernel utilizando la herramienta de creacion de perfiles nvprof.

*/

//// EJEMPLO DE DIVERGENCIA
//#include <stdio.h>
//#include <stdlib.h>
//#include <time.h>
//
//#include "cuda.h"
//#include "cuda_runtime.h"
//#include "device_launch_parameters.h"
////#include "cuda_common.cuh"
//
//// EJECUTAR EN TERMINAL******************************
//__global__ void code_without_divergence() // asignacion de valores a A y B basados en WARP
//{
//	int gid = blockIdx.x * blockDim.x + threadIdx.x;  // id de thread global
//
//	float a, b;
//	a = b = 0;
//
//	int warp_id = gid / 32;
//
//	if (warp_id % 2 == 0)
//	{
//		a = 100.0;
//		b = 50.0;
//
//
//	}
//	else
//	{
//		a = 200;
//		b = 75;
//	}
//
//}
//
//__global__ void code_with_divergence() // asignacion de valores a A y B basados en THREAD
//{
//	int gid = blockIdx.x * blockDim.x + threadIdx.x;  // id de thread global
//
//	float a, b;
//	a = b = 0;
//
//	if (gid % 2 == 0)
//	{
//		a = 100.0;
//		b = 50.0;
//
//
//	}
//	else
//	{
//		a = 200;
//		b = 75;
//	}
//
//}
//
//
//int main(int argc, char** argv)
//{
//	printf("\n---------------------------WARP DIVERGENCE EXAMPLE----------------------\n\n");
//
//	int size = 1 << 22;
//
//	dim3 block_size(128);
//	dim3 grid_size((size + block_size.x - 1) / block_size.x);
//
//	code_without_divergence << <grid_size, block_size >> > ();
//	cudaDeviceSynchronize();
//
//	code_with_divergence << <grid_size, block_size >> > ();
//	cudaDeviceSynchronize();
//
//
//	cudaDeviceReset();
//	return 0;
//}

//------------------------------------- 222 Latency ---------------------------------------

//------------------------------------- 222 Occupancy ---------------------------------------

/*

Occupancy = Active warps / maximum warps

48 registers per thread
reg_per_warp = 48 * 32 = 1536

GTX 970 device = 65536 regs per SM

Warps permitidos por SM = 65536 / 1536 = 42.67

*/

//CUDA OCCUPANCY CALCULATOR (hoja de excel)

/* GUIDE LINE FOR GRID AND BLOCK SIZE
	-keep the number of threads per block a multiple of warp size 32
	-Avoid small block sizes: Start with at least 128 or 256 threads per block
	-keep the number of blocks much greater than the number of SMs to expose sufficient parallelism to your device

 
*/