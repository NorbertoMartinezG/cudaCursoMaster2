#include "hip/hip_runtime.h"
/*
//#include "hip/hip_runtime.h"
//#include ""
//
//#include <stdio.h>
*/


//-------------------------------------219 Understadn the device better---------------------------------------

/* computer architectures classification
		-SISD - Single instruction single data
		-SIMD - single instruction multiple data
		-MISD - multiple instruction single data
		-MIMD - multiple instruction multiple data 
	CUDA se basa en SIMD
		-SIMT - single instruction multiple threads

*/


//------------------------------------- 220 Warps ---------------------------------------

/*
* -thread blocks are divide in to smaller units called warps each having 32 consecutive threads
* -warps can be defined asthe basic unit of execution in a SM
* -all threads in a warp are executed in single instrucction multiple thread (SIMT) fashion
* 
* -Los bloques se dividen en subbloques de 32
*	-ej.1. Un bloque de 128 se dividira en 4 bloques de 32 (0-31)(32-63)(64-95)(96-127)
*	-ej.2. Un bloque de 80 se dividira en 4 bloques de los cuales se dividiran (0-31)(32-39)(40-71)(72-79)
*	-ej.3. Un bloque de 1 thread activara un bloque de 32 threads de los cuales solo utilizara 1.
*/

// EJEMPLO DE USO DE BLOQUES CUANDO SE UTILIZAN 40 HILOS
//#include "hip/hip_runtime.h"
//#include ""
//
//#include <stdio.h>
//#include <stdlib.h>
//
//__global__ void print_details_of_warps()
//{
//	int gid = blockIdx.y * gridDim.x * blockDim.x + blockIdx.x * blockDim.x + threadIdx.x;
//
//	int warp_id = threadIdx.x / 32;
//
//	int gbid = blockIdx.y * gridDim.x + blockIdx.x;
//
//	printf( "tid: %d, bid.x: %d, bid.y: %d, gid: %d, warp_id: %d, gbid: %d \n",
//		threadIdx.x, blockIdx.x, blockIdx.y, gid, warp_id, gbid);
//}
//
//
//int main(int argc, char** argv) {
//
//	dim3 block_size(42);
//	dim3 grid_size(2, 2);
//
//	print_details_of_warps << <grid_size, block_size >> > ();
//	hipDeviceSynchronize();
//
//	hipDeviceReset();
//	return EXIT_SUCCESS;
//
//}

//------------------------------------- 221 Warp divergence ---------------------------------------
/*
	- forzar a algunos hilos en el warp para ejecutar diferentes instrucciones
	- la eficiencia de las ramas se puede medir para un kernel utilizando la herramienta de creacion de perfiles nvprof.

*/

//// EJEMPLO DE DIVERGENCIA
//#include <stdio.h>
//#include <stdlib.h>
//#include <time.h>
//
//#include "hip/hip_runtime.h"
//#include "hip/hip_runtime.h"
//#include ""
////#include "cuda_common.cuh"
//
//// EJECUTAR EN TERMINAL******************************
//__global__ void code_without_divergence() // asignacion de valores a A y B basados en WARP
//{
//	int gid = blockIdx.x * blockDim.x + threadIdx.x;  // id de thread global
//
//	float a, b;
//	a = b = 0;
//
//	int warp_id = gid / 32;
//
//	if (warp_id % 2 == 0)
//	{
//		a = 100.0;
//		b = 50.0;
//
//
//	}
//	else
//	{
//		a = 200;
//		b = 75;
//	}
//
//}
//
//__global__ void code_with_divergence() // asignacion de valores a A y B basados en THREAD
//{
//	int gid = blockIdx.x * blockDim.x + threadIdx.x;  // id de thread global
//
//	float a, b;
//	a = b = 0;
//
//	if (gid % 2 == 0)
//	{
//		a = 100.0;
//		b = 50.0;
//
//
//	}
//	else
//	{
//		a = 200;
//		b = 75;
//	}
//
//}
//
//
//int main(int argc, char** argv)
//{
//	printf("\n---------------------------WARP DIVERGENCE EXAMPLE----------------------\n\n");
//
//	int size = 1 << 22;
//
//	dim3 block_size(128);
//	dim3 grid_size((size + block_size.x - 1) / block_size.x);
//
//	code_without_divergence << <grid_size, block_size >> > ();
//	hipDeviceSynchronize();
//
//	code_with_divergence << <grid_size, block_size >> > ();
//	hipDeviceSynchronize();
//
//
//	hipDeviceReset();
//	return 0;
//}

//------------------------------------- 222 Latency ---------------------------------------

//------------------------------------- 223 Occupancy ---------------------------------------

/*

Occupancy = Active warps / maximum warps

48 registers per thread
reg_per_warp = 48 * 32 = 1536

GTX 970 device = 65536 regs per SM

Warps permitidos por SM = 65536 / 1536 = 42.67

*/

//CUDA OCCUPANCY CALCULATOR (hoja de excel)

/* GUIDE LINE FOR GRID AND BLOCK SIZE
	-keep the number of threads per block a multiple of warp size 32
	-Avoid small block sizes: Start with at least 128 or 256 threads per block
	-keep the number of blocks much greater than the number of SMs to expose sufficient parallelism to your device

 
*/

//------------------------------------- 224 Profiling with nvprof ---------------------------------------
/*
* sm_efficient
* achieved_occupacy
*/
//------------------------------------- 226 Parallel reduction as synchronization example----------------

/*
	-hipDeviceSynchronize -- bloquea la ejecucion de la aplicacion host hasta que las operaciones en el host esten terminadas
	- _syncthreads -- proporciona la sincronizacion en un bloque dentro del device (obliga a los hilos esperar hasta que todos los hilos lleguen a un punto

*/

//Ejemplo suma elementos de un vector  (reduccion paralela)

#include <stdio.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"
#include ""

#include "common.h"
#include "cuda_common.cuh"


__global__ void reduction_neighbored_pairs(int * input, int * temp, int size)
{
	//identificacion del hilo
	int tid = threadIdx.x;
	int gid = blockDim.x * blockIdx.x + threadIdx.x;

	if (gid>size)
	{
		return;
	}

	for (int offset = 1; offset <= blockDim.x/2; offset *=2)
	{
		if (tid % (2 * offset) == 0)
		{
			input[gid] += input[gid + offset];
		}
		__syncthreads();
	}

	if (tid == 0)
	{
		temp[blockIdx.x] = input[gid];

	}

}


int main(int argc, char** argv)
{
	printf("Running neighbored pairs reduction kernel \n");

	int size = 1 << 27; // 128 Mb of data
	int byte_size = size * sizeof(int);
	int block_size = 128;

	int* h_input, * h_ref;
	h_input = (int*)malloc(byte_size);

	initialize(h_input, size, INIT_RANDOM);

	//get the reduction result from cpu
	int cpu_result = reduction_cpu(h_input, size);

	dim3 block(block_size);
	dim3 grid(size / block.x);

	printf("kernel launch parameters | grid.x : %d, block.x : %d", grid.x, block.x);

	int temp_array_byte_size = sizeof(int) * grid.x;
	h_ref = (int*)malloc(temp_array_byte_size);

	int* d_input, * d_temp;

	gpuErrchk(hipMalloc((void**)&d_input, byte_size));
	gpuErrchk(hipMalloc((void**)&d_temp, temp_array_byte_size));

	gpuErrchk(hipMemset(d_temp, 0, temp_array_byte_size)); // establece valor inicial en 0 
	gpuErrchk(hipMemcpy(d_input, h_input, byte_size, hipMemcpyHostToDevice));

	reduction_neighbored_pairs << <grid, block >> > (d_input, d_temp, size);

	gpuErrchk(hipDeviceSynchronize());

	hipMemcpy(h_ref, d_temp, temp_array_byte_size, hipMemcpyDeviceToHost);

	int gpu_result = 0;
	for (int i = 0; i < grid.x; i++)
	{
		gpu_result += h_ref[i];
	}

	//validity check
	compare_results(gpu_result, cpu_result);

	gpuErrchk(hipFree(d_temp));
	gpuErrchk(hipFree(d_input));

	free(h_ref);
	free(h_input);


	gpuErrchk(hipDeviceReset());
	return 0;
}
