#include "hip/hip_runtime.h"
/*
//#include "hip/hip_runtime.h"
//#include ""
//
//#include <stdio.h>
*/


//-------------------------------------219 Understadn the device better---------------------------------------

/* computer architectures classification
		-SISD - Single instruction single data
		-SIMD - single instruction multiple data
		-MISD - multiple instruction single data
		-MIMD - multiple instruction multiple data 
	CUDA se basa en SIMD
		-SIMT - single instruction multiple threads

*/


//------------------------------------- 220 Warps ---------------------------------------

/*
* -thread blocks are divide in to smaller units called warps each having 32 consecutive threads
* -warps can be defined asthe basic unit of execution in a SM
* -all threads in a warp are executed in single instrucction multiple thread (SIMT) fashion
* 
* -Los bloques se dividen en subbloques de 32
*	-ej.1. Un bloque de 128 se dividira en 4 bloques de 32 (0-31)(32-63)(64-95)(96-127)
*	-ej.2. Un bloque de 80 se dividira en 4 bloques de los cuales se dividiran (0-31)(32-39)(40-71)(72-79)
*	-ej.3. Un bloque de 1 thread activara un bloque de 32 threads de los cuales solo utilizara 1.
*/

// EJEMPLO DE USO DE BLOQUES CUANDO SE UTILIZAN 40 HILOS
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

__global__ void print_details_of_warps()
{
	int gid = blockIdx.y * gridDim.x * blockDim.x + blockIdx.x * blockDim.x + threadIdx.x;

	int warp_id = threadIdx.x / 32;

	int gbid = blockIdx.y * gridDim.x + blockIdx.x;

	printf( "tid: %d, bid.x: %d, bid.y: %d, gid: %d, warp_id: %d, gbid: %d \n",
		threadIdx.x, blockIdx.x, blockIdx.y, gid, warp_id, gbid);
}


int main(int argc, char** argv) {

	dim3 block_size(42);
	dim3 grid_size(2, 2);

	print_details_of_warps << <grid_size, block_size >> > ();
	hipDeviceSynchronize();

	hipDeviceReset();
	return EXIT_SUCCESS;

}


